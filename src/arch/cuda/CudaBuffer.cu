/*
 * CudaBuffer.cpp
 *
 *  Created on: Aug 6, 2014
 *      Author: Sheng Lundquist
 */

#include "CudaBuffer.hpp"
#include "cuda_util.hpp"
#include <sys/time.h>
#include <ctime>

namespace PVCuda {

CudaBuffer::CudaBuffer(size_t inSize, hipStream_t stream)
{
   handleError(hipMalloc(&d_ptr, inSize));
   this->size = inSize;
   this->stream = stream;
}

CudaBuffer::CudaBuffer(){
   d_ptr = NULL;
   size = 0;
}

CudaBuffer::~CudaBuffer()
{
   handleError(hipFree(d_ptr));
}
   
int CudaBuffer::copyToDevice(void * h_ptr)
{
   //handleError(hipMemcpyAsync(d_ptr, h_ptr, size, hipMemcpyHostToDevice, stream));
   handleError(hipMemcpyAsync(d_ptr, h_ptr, size, hipMemcpyHostToDevice, stream));
   return 0;
}

int CudaBuffer::copyFromDevice(void * h_ptr)
{
   handleError(hipMemcpyAsync(h_ptr, d_ptr, size, hipMemcpyDeviceToHost, stream));
   return 0;
}

} // namespace PV
