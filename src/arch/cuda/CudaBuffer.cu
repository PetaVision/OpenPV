/*
 * CudaBuffer.cpp
 *
 *  Created on: Aug 6, 2014
 *      Author: Sheng Lundquist
 */

#include "CudaBuffer.hpp"
#include "cuda_util.hpp"
#include <sys/time.h>
#include <ctime>

namespace PVCuda {

CudaBuffer::CudaBuffer(size_t inSize, hipStream_t stream)
{
   handleError(hipMalloc(&d_ptr, inSize));
   this->size = inSize;
   this->stream = stream;
}

CudaBuffer::CudaBuffer(){
   d_ptr = NULL;
   size = 0;
}

CudaBuffer::~CudaBuffer()
{
   handleError(hipFree(d_ptr));
}
   
int CudaBuffer::copyToDevice(void * h_ptr)
{
   //handleError(hipMemcpyAsync(d_ptr, h_ptr, size, hipMemcpyHostToDevice, stream));
   handleError(hipMemcpyAsync(d_ptr, h_ptr, size, hipMemcpyHostToDevice, stream));
   return 0;
}
   
///**
// * Convert to milliseconds
// */
//long get_cpu_time() {
//   struct timeval tim;
//   //   struct rusage ru;
//   //   getrusage(RUSAGE_SELF, &ru);
//   //   tim = ru.ru_utime;
//   gettimeofday(&tim, NULL);
//   //printf("get_cpu_time: sec==%d usec==%d\n", tim.tv_sec, tim.tv_usec);
//   return ((long) tim.tv_sec)*1000000 + (long) tim.tv_usec;
//}

int CudaBuffer::copyFromDevice(void * h_ptr)
{
   //hipEvent_t eStart, eStop;
   //float gpuTime;
   //hipEventCreate(&eStart);
   //hipEventCreate(&eStop);

   //hipDeviceSynchronize();

   //hipEventRecord(eStart, stream);
   //long start = get_cpu_time();
   handleError(hipMemcpyAsync(h_ptr, d_ptr, size, hipMemcpyDeviceToHost, stream));
   //hipMemcpy(h_ptr, d_ptr, size, hipMemcpyDeviceToHost);
   //long stop = get_cpu_time();
   //hipEventRecord(eStop, stream);
   //hipEventSynchronize(eStop);
   //hipEventElapsedTime(&gpuTime, eStart, eStop);
   //printf("cpu run time: %f\n", (double)(stop-start)/1000);
   //printf("gpu run time: %f\n", gpuTime);
   return 0;
}

} // namespace PV
