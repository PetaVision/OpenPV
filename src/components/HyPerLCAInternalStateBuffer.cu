#include "hip/hip_runtime.h"
#include "HyPerLCAInternalStateBuffer.hpp"
#include "cudakernels/conversions.hcu"

namespace PVCuda {

__global__ void updateKernelHyPerLCAInternalStateBuffer(
      int const nbatch,
      int const numNeurons,
      int const nx,
      int const ny,
      int const nf,
      int const lt,
      int const rt,
      int const dn,
      int const up,
      float *V,
      bool const selfInteract,
      double const *dtAdapt,
      float const tau,
      float const *accumulatedGSyn,
      float const *A) {

   int n = (blockIdx.x * blockDim.x) + threadIdx.x;
   if (n < numNeurons * nbatch) {
      int b = n / numNeurons; // batch index
      int k = n % numNeurons; // neuron index within batch element

      float exp_tau                     = (float)std::exp(-dtAdapt[b] / (double)tau);
      float *VBatch                     = V + b * numNeurons;
      float const *accumulatedGSynBatch = accumulatedGSyn + b * numNeurons;
      float const gSyn                  = accumulatedGSynBatch[k];
      // Activity is an extended buffer.
      float const *ABatch = A + b * (nx + rt + lt) * (ny + up + dn) * nf;

      int kex   = kIndexExtended(k, nx, ny, nf, lt, rt, dn, up);
      VBatch[k] = exp_tau * VBatch[k] + (1.0f - exp_tau) * (gSyn + selfInteract * ABatch[kex]);
   }
}

} // end namespace PVCuda

namespace PV {

void HyPerLCAInternalStateBuffer::runKernel() {
   PVLayerLoc const *loc           = getLayerLoc();
   int const nx                    = loc->nx;
   int const ny                    = loc->ny;
   int const nf                    = loc->nf;
   int const lt                    = loc->halo.lt;
   int const rt                    = loc->halo.rt;
   int const dn                    = loc->halo.dn;
   int const up                    = loc->halo.up;
   int const numNeurons            = nx * ny * nf;
   int const nbatch                = loc->nbatch;
   int const numNeuronsAcrossBatch = numNeurons * nbatch;
   int currBlockSize               = mCudaDevice->get_max_threads();
   hipStream_t cudaStream         = mCudaDevice->getStream();
   // Ceil to get all weights
   int currGridSize = (int)ceil(((float)numNeuronsAcrossBatch) / currBlockSize);
   // Call function
   PVCuda::updateKernelHyPerLCAInternalStateBuffer<<<currGridSize, currBlockSize, 0, cudaStream>>>(
         nbatch,
         numNeurons,
         nx,
         ny,
         nf,
         lt,
         rt,
         dn,
         up,
         (float *)getCudaBuffer()->getPointer(),
         mSelfInteract,
         (double const *)mCudaDtAdapt->getPointer(),
         mScaledTimeConstantTau,
         (float const *)mAccumulatedGSyn->getCudaBuffer()->getPointer(),
         (float const *)mActivity->getCudaBuffer()->getPointer());
}

} // end namespace PV
