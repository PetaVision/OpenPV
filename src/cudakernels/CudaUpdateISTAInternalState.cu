#include "hip/hip_runtime.h"
#include "CudaUpdateISTAInternalState.hpp"
#include "conversions.hcu"

namespace PVCuda {

__global__ void HyPerLCAInternalState_update_state(
      int const nbatch,
      int const numNeurons,
      int const nx,
      int const ny,
      int const nf,
      int const lt,
      int const rt,
      int const dn,
      int const up,
      float *V,
      float const VThresh,
      double *dtAdapt,
      float const tau,
      float const *accumulatedGSyn,
      float const *A) {

   int kIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
   if (kIndex < numNeurons * nbatch) {
      int b                             = kIndex / numNeurons;
      int k                             = kIndex % numNeurons;
      float *VBatch                     = V + b * numNeurons;
      float const *accumulatedGSynBatch = accumulatedGSyn + b * numNeurons;
      // Activity is an extended buffer.
      float const *ABatch = A + b * (nx + rt + lt) * (ny + up + dn) * nf;

      float const gSyn = accumulatedGSynBatch[k];
      int kex          = kIndexExtended(k, nx, ny, nf, lt, rt, dn, up);
      float sign       = 0.0f;
      if (ABatch[kex] != 0.0f) {
         sign = ABatch[kex] / fabsf(ABatch[kex]);
      }
      VBatch[k] += ((float)dtAdapt[b] / tau) * (gSyn - (VThresh * sign));
   }
}

int CudaUpdateISTAInternalState::do_run() {
   int currBlockSize = device->get_max_threads();
   // Ceil to get all weights
   int currGridSize = ceil(((float)params.numNeurons * params.nbatch) / currBlockSize);
   // Call function
   HyPerLCAInternalState_update_state<<<currGridSize, currBlockSize, 0, device->getStream()>>>(
         params.nbatch,
         params.numNeurons,
         params.nx,
         params.ny,
         params.nf,
         params.lt,
         params.rt,
         params.dn,
         params.up,
         params.V,
         params.VThresh,
         params.dtAdapt,
         params.tau,
         params.accumulatedGSyn,
         params.activity);
   return 0;
}

} // end namespace PVCuda
