#include "hip/hip_runtime.h"
#include "CudaRecvPre.hpp"
#include "../arch/cuda/cuda_util.hpp"
#include "conversions.hcu"

namespace PVCuda{

//Kernel code
__global__
void HyPerLayer_recv_pre(
   recv_pre_params params,
   int batchIdx
){
   unsigned int kPreExt;
   float a;
   PVPatch patch;
   int wIdx;
   int numberShrunkenWeights;

   long tIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

   //Put this on cpu
   int fullPatchSize = params.nfp * params.nxp * params.nyp;

   if(params.isSparse){
      if(tIndex >= fullPatchSize * params.numActive[batchIdx]){
         return;
      }
   }
   else{
      if(tIndex >= fullPatchSize * params.numPreExt){
         return;
      }
   }

   unsigned int neuronIndex = tIndex / fullPatchSize;

   int preBatchOffset = batchIdx * params.numPreExt;
   if(params.isSparse){
      kPreExt = params.activeIndices[neuronIndex + preBatchOffset];
   }
   else{
      kPreExt = neuronIndex;
   }
   a = params.preData[kPreExt + preBatchOffset] * params.dt_factor;
   int kernelIndex;
   if(params.sharedWeights == 1){
      kernelIndex = params.patch2datalookuptable[kPreExt];
   }
   else{
      kernelIndex = kPreExt;
   }
   //Grab weight patches
   patch = params.patches[kPreExt];
   wIdx = kernelIndex * fullPatchSize + patch.offset;
   numberShrunkenWeights = params.nfp * patch.nx * patch.ny;

   //__syncthreads();

   if(a == 0) return;
   //patch may be shrunken, if thread oob, return
   int patchIndex = tIndex % fullPatchSize;
   if(patchIndex >= numberShrunkenWeights){
      return;
   }

   int postBatchOffset = batchIdx * params.numPostRes;
   float* gSynStart = params.postGSyn + postBatchOffset + params.gSynPatchStart[kPreExt];

   //Calculate what y row patchIndex is in
   int ky = kyPos(patchIndex, patch.nx, patch.ny, params.nfp);
   int kx = kxPos(patchIndex, patch.nx, patch.ny, params.nfp); 
   int kf = featureIndex(patchIndex, patch.nx, patch.ny, params.nfp); 
   int k = kx * params.nfp + kf;

   float * gSynPtr = gSynStart + ky*params.sy + k;
   float weightVal = params.weights[wIdx + ky*params.syw + k];

   //Multiply values
   float outVal = a * weightVal;

   //Atomic add into postGSyn
   atomicAdd(gSynPtr, outVal);
}


CudaRecvPre::CudaRecvPre(CudaDevice* inDevice):CudaKernel(inDevice){
}

CudaRecvPre::~CudaRecvPre(){
}

void CudaRecvPre::setArgs(
      int nbatch,
      int numPreExt,
      int numPostRes,
      int nxp,
      int nyp,
      int nfp,

      int sy,
      int syw,
      float dt_factor,
      int sharedWeights,

      /* PVPatch* */ CudaBuffer* patches,
      /* size_t* */  CudaBuffer* gSynPatchStart,

      /* float* */   CudaBuffer* preData,
      /* float* */   CudaBuffer* weights,
      /* float* */   CudaBuffer* postGSyn,
      /* int* */     CudaBuffer* patch2datalookuptable,

      bool isSparse,
      /*unsigned long*/ CudaBuffer* numActive,
      /*unsigned int*/ CudaBuffer* activeIndices
   ){
   params.nbatch = nbatch;
   params.numPreExt = numPreExt;
   params.numPostRes = numPostRes;

   params.nxp = nxp;
   params.nyp = nyp;
   params.nfp = nfp;

   params.sy = sy;
   params.syw = syw;
   params.dt_factor = dt_factor;
   params.sharedWeights = sharedWeights;

   params.patches = (PVPatch*)patches->getPointer();
   params.gSynPatchStart = (size_t*)gSynPatchStart->getPointer();

   params.preData = (float*)preData->getPointer();
   params.weights = (float*)weights->getPointer();
   params.postGSyn = (float*)postGSyn->getPointer();
   params.patch2datalookuptable = (int*)patch2datalookuptable->getPointer();

   params.isSparse = isSparse;
   if(activeIndices){
      params.numActive = (long*)numActive->getPointer();
      params.activeIndices = (unsigned int*)activeIndices->getPointer();
   }
   else{
      params.activeIndices = NULL;
      params.numActive = NULL;
   }

   setArgsFlag();
}

int CudaRecvPre::do_run(){

   size_t sharedSize = 0;

   if(sharedSize > device->get_local_mem()){
      printf("run: given shared memory size of %zu is bigger than allowed shared memory size of %zu\n", sharedSize, device->get_local_mem());
      exit(-1);
   }

   for(int b = 0; b < params.nbatch; b++){
      HyPerLayer_recv_pre<<<grid_size, block_size, sharedSize>>>(
         params,
         b
      );
   }

   handleCallError("Cuda recv pre run");

   return 0;
}

}
