#include "hip/hip_runtime.h"
#include "CudaRecvPost.hpp"
#include "../arch/cuda/cuda_util.hpp"
#include "../arch/cuda/device_util.hpp"

namespace PVCuda{

//Kernel code
__global__
void HyPerLayer_recv_post(recv_post_params params){
   ////Shared memory buffers are declared
   extern __shared__ char sharedMem[];
   //__shared__ float* preBuffer;
   __shared__ float* postBuffer;
   __shared__ float* weightsBuffer;
   //__shared__ long* localStartSourceExt;
   //preBuffer = (float*)sharedMem;
   //postBuffer = (float*)(&(preBuffer[params.preBufNum]));
   postBuffer = (float*)sharedMem;
   weightsBuffer = (float*)(&(postBuffer[params.postBufNum]));

   //Ordered this way because threads vary fastest in x, then y, then z
   //Mapped to petavision order of f, x, and y

   int localF = blockDim.x;
   int localX = blockDim.y;
   int localY = blockDim.z;
   
   int localFIndex = threadIdx.x;
   int localXIndex = threadIdx.y;
   int localYIndex = threadIdx.z;

   int fTargetRes = (blockIdx.x * blockDim.x) + threadIdx.x;
   int xTargetRes = (blockIdx.y * blockDim.y) + threadIdx.y;
   int yTargetRes = (blockIdx.z * blockDim.z) + threadIdx.z;

   ////Calculate kTargetRes based on x, y, and f
   int kTargetRes = kIndex(xTargetRes, yTargetRes, fTargetRes, params.nxRes, params.nyRes, params.nf);

   int kTargetExt = kIndexExtended(kTargetRes, params.nxRes, params.nyRes, params.nf, params.nblt, params.nbrt, params.nbdn, params.nbup);

   //Each wIdx should be shared since each workgroup convolves one weight kernel
   __shared__ int wIdx;
   if(localXIndex == 0 && localYIndex == 0){
      //Change restricted to extended post neuron
      int kernelIndex;
      if(params.sharedWeights == 1){
         kernelIndex = params.patch2datalookuptable[kTargetExt];
      }
      else{
         kernelIndex = kTargetExt;
      }
      wIdx = kernelIndex * params.nxp * params.nyp * params.nfp;
   }

   //Get top left most neuron in the group
   //__shared__ long localStartSourceExt;
   //if(localXIndex == 0 && localYIndex == 0 && localFIndex == 0){
   //   localStartSourceExt = params.startSourceExtBuf[kTargetRes];
   //}

   long startSourceExt = params.startSourceExtBuf[kTargetRes];

   int localIndex = kIndex(localXIndex, localYIndex, localFIndex, localX, localY, localF);

   postBuffer[localIndex] = 0;
      
   int numXfBuffer = params.localBufSizeX * params.nfp;
   int numWeightsBuffer = params.nxp * params.nfp;

   int xOffset = localXIndex * params.preToPostScaleX;
   //int yOffset = localYIndex * params.preToPostScaleY;

   //Wait for shared memory loads
   __syncthreads();

   for(int ky = 0; ky < params.nyp; ky++){
      //Copy global to local, do this with all threads
      //Pre buffer
      //if(localIndex < warpSize){
      //   for(int i = localIndex; i < numXfBuffer; i+= warpSize){
      //      preBuffer[i] = params.preData[localStartSourceExt + ky * params.sy + i];
      //   }
      //}

      //Weights
      if(localIndex < warpSize){
         for(int i = localIndex; i < numWeightsBuffer; i+= warpSize){
            weightsBuffer[i] = params.weights[wIdx + ky * params.syp + i];
         }
      }
      //The actual pre buffer index
      __syncthreads();

      float* activityY = &(params.preData[startSourceExt + ky * params.sy]);
      //float* activityY = &(preBuffer[xOffset * params.nfp]);
      //float* activityY = &(preBuffer[(ky+yOffset) * params.localBufSizeX * params.nfp + xOffset*params.nfp]);

      float* weightY = weightsBuffer;
      //float* weightY = &(params.weights[wIdx + ky * params.syp]);
      //pvpatch_accumulate_from_post(numPerStride, postAddr, activityY, weightY, dt_factor, (void*)0);

      //Summing into post buffer indexed by localIndex
      int k;
      for (k = 0; k < params.numPerStride; k++) {
         postBuffer[localIndex] += activityY[k]*weightY[k]*params.dt_factor;
         //postBuffer[localIndex] += activityY[k]*weightsBuffer[k]*params.dt_factor;
      }
      __syncthreads();
   }

   ////Sum into global memory
   params.postGsyn[kTargetRes] += postBuffer[localIndex];
}


CudaRecvPost::CudaRecvPost(CudaDevice* inDevice):CudaKernel(inDevice){
}

CudaRecvPost::~CudaRecvPost(){
}

void CudaRecvPost::setArgs(
      const int nxRes, //num post neurons
      const int nyRes,
      const int nf,

      const int nblt, //Border of orig
      const int nbrt, //Border of orig
      const int nbdn, //Border of orig
      const int nbup, //Border of orig

      const int nxp,
      const int nyp,
      const int nfp,

      const int localBufSizeX,
      const int localBufSizeY,
      const float preToPostScaleX,
      const float preToPostScaleY,

      const int sy,
      const int syp,
      const int numPerStride,
      const float dt_factor,
      const int sharedWeights,

      /* long* */  CudaBuffer* startSourceExtBuf,
      /* float* */ CudaBuffer* preData,
      /* float* */ CudaBuffer* weights,
      /* float* */ CudaBuffer* postGsyn,
      /* int* */   CudaBuffer* patch2datalookuptable
   ){
   params.nxRes = nxRes;
   params.nyRes = nyRes;
   params.nf = nf;

   params.nblt = nblt;
   params.nbrt = nbrt;
   params.nbdn = nbdn;
   params.nbup = nbup;

   params.nxp = nxp;
   params.nyp = nyp;
   params.nfp = nfp;

   params.localBufSizeX = localBufSizeX;
   params.localBufSizeY = localBufSizeY;
   params.preToPostScaleX = preToPostScaleX;
   params.preToPostScaleY = preToPostScaleY;

   params.sy = sy;
   params.syp = syp;
   params.numPerStride = numPerStride;
   params.dt_factor = dt_factor;
   params.sharedWeights = sharedWeights;

   params.startSourceExtBuf = (long*)startSourceExtBuf->getPointer();
   params.preData = (float*)preData->getPointer();
   params.weights = (float*)weights->getPointer();
   params.postGsyn = (float*)postGsyn->getPointer();
   params.patch2datalookuptable = (int*)patch2datalookuptable->getPointer();

   params.warpSize = device->get_warp_size();

   setArgsFlag();
}

int CudaRecvPost::run(){
   
   params.postBufNum = block_size.x * block_size.y * block_size.z;

   //int singlePreBufNum = params.localBufSizeX * params.nfp;
   //int singleWeightsBufNum = params.nxp * params.nfp;
   //params.numXfBufs = floor((device->get_local_mem()-sizeof(float)*params.postBufNum)/((singlePreBufNum + singleWeightsBufNum) * sizeof(float)));

   //params.numXfBufs = params.numXfBufs < params.nyp ? params.numXfBufs : params.nyp;  
   //params.numXfBufs = 1;

   //params.preBufNum = params.localBufSizeX * params.nfp;
   params.weightsBufNum = params.nxp * params.nfp;

   //size_t sharedSize = sizeof(float) * (params.preBufNum + params.postBufNum + params.weightsBufNum);
   size_t sharedSize = sizeof(float) * (params.postBufNum + params.weightsBufNum);

   if(sharedSize > device->get_local_mem()){
      printf("gpu post run: given shared memory size of %zu is bigger than allowed shared memory size of %zu\n", sharedSize, device->get_local_mem());
      exit(-1);
   }

   ////If sharedSize is greater than device's local memory, then numXFBufs should be greater than 1
   //assert(params.numXfBufs >= 1);

   if(block_size.x != 1){
      printf("gpu post run: numFLocal must be 1\n");
      exit(-1);
   }
   //if(block_size.z != 1){
   //   printf("gpu post run: numYLocal must be 1\n");
   //   exit(-1);
   //}

   //printf("Using %d buffers\n", params.numXfBufs);
   
   HyPerLayer_recv_post<<<grid_size, block_size, sharedSize>>>(params);
   handleCallError();

   return 0;
}

}
