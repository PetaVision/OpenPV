#include "hip/hip_runtime.h"
#include "CudaRecvPost.hpp"
#include "conversions.hcu"

namespace PVCuda{

#ifdef PV_USE_CUDNN
#include <hipDNN.h>

//Function to change PV representation to CUDNN representation
//Does 2 things: permutate ordering from [outFeature, ny, nx, inFeature] to [outFeature, inFeature, ny, nx]
//Reshapes the matrix if manyScale > 1 to map different "many" kernels into feature dimension
//Coalesced in input
__global__
void CudaPermutePVToCudnn(float* dest, float* src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY, int cropX, int cropY){
   //parameter dimensions are in source PV format
   int destNx = (nx-2*cropX)/manyScaleX;
   int destNy = (ny-2*cropY)/manyScaleY;
   int destInFeatures = inFeatures*manyScaleX*manyScaleY;

   int kSrc = (blockIdx.x * blockDim.x) + threadIdx.x;
   if(kSrc < outFeatures * ny * nx * inFeatures){
      int kOF = kSrc/(ny*nx*inFeatures);
      int kY  = (kSrc % (ny*nx*inFeatures))/(nx*inFeatures);
      int kX  = (kSrc % (nx*inFeatures))/inFeatures;
      int kIF = (kSrc % inFeatures);

      //check if in bounds
      if(kX < cropX || kX >= nx-cropX){ 
         return;
      }
      else{
         kX = kX - cropX;
      }
      if(kY < cropY || kY >= ny-cropY){
         return;
      }
      else{
         kY = kY - cropY;
      }

      //Recalculate x, y, and f based on manyScale
      kIF = kIF + inFeatures * (kX % manyScaleX + (kY % manyScaleY) * manyScaleX);
      kX = kX/manyScaleX;
      kY = kY/manyScaleY;

      int sOF = destInFeatures * destNy * destNx;
      int sIF = destNy * destNx;
      int sY  = destNx;

      int kDest = kOF * sOF + kIF * sIF + kY * sY + kX;

      dest[kDest] = src[kSrc];
   }
}

//Weights need to be reversed for cudnn
//No need to account for many because the PV representation matches with how gsyn was reshaped.
__global__
void CudaPermuteWeightsPVToCudnn(float* dest, float* src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY){
   //Parameter dimensions are PV source dimensions
   int kSrc = (blockIdx.x * blockDim.x) + threadIdx.x;
   if(kSrc < outFeatures * manyScaleX * manyScaleY * ny * nx * inFeatures){
      int kOF = kSrc/(ny*nx*inFeatures);
      int kY  = (kSrc % (ny*nx*inFeatures))/(nx*inFeatures);
      int kX  = (kSrc % (nx*inFeatures))/inFeatures;
      int kIF = (kSrc % inFeatures);

      int sOF = inFeatures * ny * nx;
      int sIF = ny * nx;
      int sY  = nx;

      int kDest = kOF * sOF + kIF * sIF + (ny-kY-1) * sY + (nx-kX-1);

      dest[kDest] = src[kSrc];
   }
}

__global__
void CudaPermuteCudnnToPV(float* dest, float* src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY){
   //parameter dimensions are in dest PV format
   int srcNx = nx/manyScaleX;
   int srcNy = ny/manyScaleY;
   int srcInFeatures = inFeatures*manyScaleX*manyScaleY;

   int kDest = (blockIdx.x * blockDim.x) + threadIdx.x;
   if(kDest < outFeatures * ny * nx * inFeatures){
      int kOF = kDest/(ny*nx*inFeatures);
      int kY  = (kDest % (ny*nx*inFeatures))/(nx*inFeatures);
      int kX  = (kDest % (nx*inFeatures))/inFeatures;
      int kIF = (kDest % inFeatures);

      //Recalculate x, y, and f based on manyScale
      kIF = kIF + inFeatures * (kX % manyScaleX + (kY % manyScaleY) * manyScaleX);
      kX = kX/manyScaleX;
      kY = kY/manyScaleY;

      int sOF = srcInFeatures * srcNy * srcNx;
      int sIF = srcNy * srcNx;
      int sY  = srcNx;

      int kSrc = kOF * sOF + kIF * sIF + kY * sY + kX;

      dest[kDest] = src[kSrc];
   }
}

#endif // PV_USE_CUDNN

//Kernel code
__global__
void HyPerLayer_recv_post(recv_post_params params, int batch){
   ////Shared memory buffers are declared
   extern __shared__ char sharedMem[];
   __shared__ float* preBuffer;
   __shared__ float* postBuffer;
   __shared__ float* weightsBuffer;

   postBuffer = (float*)sharedMem;
   weightsBuffer = (float*)(&(postBuffer[params.postBufNum]));

   if(params.preDataLocal){
      preBuffer = (float*)(&(weightsBuffer[params.weightsBufNum]));
   }

   //Ordered this way because threads vary fastest in x, then y, then z
   //Mapped to petavision order of f, x, and y

   int localF = blockDim.x;
   int localX = blockDim.y;
   int localY = blockDim.z;
   
   int localFIndex = threadIdx.x;
   int localXIndex = threadIdx.y;
   int localYIndex = threadIdx.z;

   int fTargetRes = (blockIdx.x * blockDim.x) + threadIdx.x;
   int xTargetRes = (blockIdx.y * blockDim.y) + threadIdx.y;
   int yTargetRes = (blockIdx.z * blockDim.z) + threadIdx.z;

   ////Calculate kTargetRes based on x, y, and f
   int kTargetRes = kIndex(xTargetRes, yTargetRes, fTargetRes, params.nxRes, params.nyRes, params.nf);

   int kTargetExt = kIndexExtended(kTargetRes, params.nxRes, params.nyRes, params.nf, params.nblt, params.nbrt, params.nbdn, params.nbup);

   //Each wIdx should be shared since each workgroup convolves one weight kernel
   __shared__ int wIdx;
   if(localXIndex == 0 && localYIndex == 0){
      //Change restricted to extended post neuron
      int kernelIndex;
      if(params.sharedWeights == 1){
         kernelIndex = params.patch2datalookuptable[kTargetExt];
      }
      else{
         kernelIndex = kTargetExt;
      }
      wIdx = kernelIndex * params.nxp * params.nyp * params.nfp;
   }

   //Get top left most neuron in the group
   __shared__ long localStartSourceExt;
   long startSourceExt;
   if(params.preDataLocal){
      if(localXIndex == 0 && localYIndex == 0 && localFIndex == 0){
         localStartSourceExt = params.startSourceExtBuf[kTargetRes];
      }
   }
   else{
      startSourceExt = params.startSourceExtBuf[kTargetRes];
   }

   int localIndex = kIndex(localXIndex, localYIndex, localFIndex, localX, localY, localF);

   postBuffer[localIndex] = 0;
      
   int numXfBuffer = params.localBufSizeX * params.nfp;
   int numWeightsBuffer = params.nxp * params.nfp;

   int xOffset = localXIndex * params.preToPostScaleX;
   //int yOffset = localYIndex * params.preToPostScaleY;

   int numCopyThreads = localF * localX * localY < warpSize ? localF * localX * localY : warpSize;
   
   //Wait for shared memory loads
   __syncthreads();

   int preBatchOffset = batch * (params.preNx + params.preNblt + params.preNbrt) * (params.preNy + params.preNbup + params.preNbdn) * params.preNf; 

   for(int ky = 0; ky < params.nyp; ky++){
      //Copy global to local, do this with all threads
      if(params.preDataLocal){
         //Pre buffer
         if(localIndex < numCopyThreads){
            for(int i = localIndex; i < numXfBuffer; i+= numCopyThreads){
               preBuffer[i] = params.preData[preBatchOffset + localStartSourceExt + ky * params.sy + i];
            }
         }
      }

      //Weights
      if(localIndex < numCopyThreads){
         for(int i = localIndex; i < numWeightsBuffer; i+= numCopyThreads){
            weightsBuffer[i] = params.weights[wIdx + ky * params.syp + i];
         }
      }
      //The actual pre buffer index
      __syncthreads();

      float* activityY;
      if(params.preDataLocal){
         activityY = &(preBuffer[xOffset * params.nfp]);
      }
      else{
         activityY = &(params.preData[preBatchOffset + startSourceExt + ky * params.sy]);
      }

      float* weightY = weightsBuffer;
      //float* weightY = &(params.weights[wIdx + ky * params.syp]);

      //Summing into post buffer indexed by localIndex
      int k;
      for (k = 0; k < params.numPerStride; k++) {
         postBuffer[localIndex] += activityY[k]*weightY[k]*params.dt_factor;
      }
      __syncthreads();
   }
   ////Sum into global memory
   int postBatchOffset = batch * params.nxRes * params.nyRes * params.nf; 
   params.postGsyn[postBatchOffset + kTargetRes] += postBuffer[localIndex];
}

#ifdef PV_USE_CUDNN
void CudaRecvPost::callPermuteDatastorePVToCudnnKernel(int gridSize, int blockSize, int nbatch, int ny, int nx, int nf) {
   //Datastore will never get reshaped, so manyScale will always be 1
   CudaPermutePVToCudnn<<<gridSize, blockSize, 0, device->getStream()>>>(params.cudnn_preData, params.preData, nbatch, ny, nx, nf, 1, 1, params.diffX, params.diffY);
}

void CudaRecvPost::callPermuteGSynPVToCudnnKernel(int gridSize, int blockSize, float* gSynPatchHead, int nbatch, int ny, int nx, int nf) {
   CudaPermutePVToCudnn<<<gridSize, blockSize, 0, device->getStream()>>>(params.cudnn_gSyn, gSynPatchHead, nbatch, ny, nx, nf, params.manyScaleX, params.manyScaleY, 0, 0);
}

void CudaRecvPost::callPermuteGSynCudnnToPVKernel(int gridSize, int blockSize, float* gSynPatchHead, int nbatch, int ny, int nx, int nf) {
   CudaPermuteCudnnToPV<<<gridSize, blockSize, 0, device->getStream()>>>(gSynPatchHead, params.cudnn_gSyn, nbatch, ny, nx, nf, params.manyScaleX, params.manyScaleY);
}

#endif // PV_USE_CUDNN

}  // end namespace PVCuda
