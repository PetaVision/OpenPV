#include "hip/hip_runtime.h"
#include "CudaUpdateStateFunctions.hpp"
#include "../arch/cuda/cuda_util.hpp"
#include "conversions.hcu"



namespace PVCuda{
//Include update state functions with cuda flag on 
#include "../layers/updateStateFunctions.h"

//The actual wrapper kernel code thats calling updatestatefunctions
__global__
void HyPerLCALayer_update_state(
				const int nbatch,
				const int numNeurons,
				const int nx,
				const int ny,
				const int nf,
				const int lt,
				const int rt,
				const int dn,
				const int up,
				const int numChannels,
				float * V,
				const int numVertices,
				float * verticesV,
				float * verticesA,
				float * slopes,
				const bool selfInteract,
				double * dtAdapt,
				const float tau,
				float * GSynHead,
				float * activity)
{

   if((blockIdx.x * blockDim.x) + threadIdx.x < numNeurons*nbatch){
      updateV_HyPerLCALayer(
			    nbatch,
			    numNeurons,
			    numChannels,
			    V,
			    GSynHead,
			    activity,
			    numVertices,
			    verticesV,
			    verticesA,
			    slopes,
			    dtAdapt,
			    tau,
			    selfInteract,
			    nx,
			    ny,
			    nf,
			    lt,
			    rt,
			    dn,
			    up);
   }
}
__global__
void ISTALayer_update_state(
			    const int nbatch,
			    const int numNeurons,
			    const int nx,
			    const int ny,
			    const int nf,
			    const int lt,
			    const int rt,
			    const int dn,
			    const int up,
			    const int numChannels,
			    float * V,
			    const float Vth,
			    double * dtAdapt,
			    const float tau,
			    float * GSynHead,
			    float * activity)
  {
    if((blockIdx.x * blockDim.x) + threadIdx.x < numNeurons*nbatch){
      updateV_ISTALayer(nbatch, 
			numNeurons, 
			V, 
			GSynHead, 
			activity,
			Vth, 
			dtAdapt, 
			tau, 
			nx, 
			ny, 
			nf, 
			lt, 
			rt, 
			dn, 
			up, 
			numChannels);
    }
  }

CudaUpdateHyPerLCALayer::CudaUpdateHyPerLCALayer(CudaDevice* inDevice):CudaKernel(inDevice){
}
  
CudaUpdateHyPerLCALayer::~CudaUpdateHyPerLCALayer(){
}

CudaUpdateISTALayer::CudaUpdateISTALayer(CudaDevice* inDevice):CudaKernel(inDevice){
}

CudaUpdateISTALayer::~CudaUpdateISTALayer(){
}

void CudaUpdateHyPerLCALayer::setArgs(
				      const int nbatch,
				      const int numNeurons,
				      const int nx,
				      const int ny,
				      const int nf,
				      const int lt,
				      const int rt,
				      const int dn,
				      const int up,
				      const int numChannels,
				      
				      /* float* */ CudaBuffer* V,
				      
				      const int numVertices,
				      /* float* */ CudaBuffer* verticesV,
				      /* float* */ CudaBuffer* verticesA,
				      /* float* */ CudaBuffer* slopes,
				      const bool selfInteract,
				      /* double* */ CudaBuffer* dtAdapt,
				      const float tau,
				      
				      /* float* */ CudaBuffer* GSynHead,
				      /* float* */ CudaBuffer* activity
				      ){
  params.nbatch = nbatch;
  params.numNeurons = numNeurons;
  params.nx = nx;
  params.ny = ny;
  params.nf = nf;
  params.lt = lt;
  params.rt = rt;
  params.dn = dn;
  params.up = up;
  params.numChannels = numChannels;
  
  params.V = (float*) V->getPointer();
  
   params.numVertices = numVertices;
   params.verticesV = (float*) verticesV->getPointer();
   params.verticesA = (float*) verticesA->getPointer();
   params.slopes = (float*) slopes->getPointer();
   params.selfInteract = selfInteract;
   params.dtAdapt = (double*) dtAdapt->getPointer();
   params.tau = tau;
   
   params.GSynHead = (float*) GSynHead->getPointer();
   params.activity = (float*) activity->getPointer();
   
   setArgsFlag();
}


int CudaUpdateHyPerLCALayer::do_run(){
   int currBlockSize = device->get_max_threads();
   //Ceil to get all weights
   int currGridSize = ceil(((float)params.numNeurons * params.nbatch)/currBlockSize);
   //Call function
   HyPerLCALayer_update_state<<<currGridSize, currBlockSize, 0, device->getStream()>>>(
   params.nbatch,
   params.numNeurons,
   params.nx,
   params.ny,
   params.nf,
   params.lt,
   params.rt,
   params.dn,
   params.up,
   params.numChannels,
   params.V,
   params.numVertices,
   params.verticesV,
   params.verticesA,
   params.slopes,
   params.selfInteract,
   params.dtAdapt,
   params.tau,
   params.GSynHead,
   params.activity);
   handleCallError("HyPerLCALayer Update run");
   return 0;
}

void CudaUpdateISTALayer::setArgs(
				  const int nbatch,
				  const int numNeurons,
				  const int nx,
				  const int ny,
				  const int nf,
				  const int lt,
				  const int rt,
				  const int dn,
				  const int up,
				  const int numChannels,
				  
				  /* float* */ CudaBuffer* V,
				  
				  const float Vth,
				  /* double* */ CudaBuffer* dtAdapt,
				  const float tau,
				  
				  /* float* */ CudaBuffer* GSynHead,
				  /* float* */ CudaBuffer* activity
				  ){
  params.nbatch = nbatch;
  params.numNeurons = numNeurons;
  params.nx = nx;
  params.ny = ny;
  params.nf = nf;
  params.lt = lt;
  params.rt = rt;
  params.dn = dn;
  params.up = up;
  params.numChannels = numChannels;
  
  params.V = (float*) V->getPointer();
  
  params.Vth = Vth;
  params.dtAdapt = (double*) dtAdapt->getPointer();
  params.tau = tau;
  
  params.GSynHead = (float*) GSynHead->getPointer();
  params.activity = (float*) activity->getPointer();
    
  setArgsFlag();
}

int CudaUpdateISTALayer::do_run(){
    int currBlockSize = device->get_max_threads();
    //Ceil to get all weights                                                                                           
    int currGridSize = ceil(((float)params.numNeurons * params.nbatch)/currBlockSize);
    //Call function
    ISTALayer_update_state<<<currGridSize, currBlockSize, 0, device->getStream()>>>(
    params.nbatch,
    params.numNeurons,
    params.nx,
    params.ny,
    params.nf,
    params.lt,
    params.rt,
    params.dn,
    params.up,
    params.numChannels,
    params.V,
    params.Vth,
    params.dtAdapt,
    params.tau,
    params.GSynHead,
    params.activity);
    handleCallError("ISTALayer Update run");
    return 0;
}

}
